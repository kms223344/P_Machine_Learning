#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__device__ int addem( int a, int b ) {
    return a + b;
}

__global__ void add( int a, int b, int *c ) {
    *c = addem( a, b );
}

int main( void ) {
    int c;
    int *dev_c;
   hipMalloc( (void**)&dev_c, sizeof(int) ) ;

    add<<<1,1>>>( 2, 7, dev_c );

   HANDLE_ERROR( hipMemcpy( &c, dev_c, sizeof(int),
                              hipMemcpyDeviceToHost ) );
    printf( "2 + 7 = %d\n", c );
    HANDLE_ERROR( hipFree( dev_c ) );

    return 0;
}