#include "hip/hip_runtime.h"
#include "mat_sqrt_d_kernel.h"

#define BLOCK_SIZE 32

__device__ __forceinline__ float mat_sqrt_d (float a, float alpha){
    //return 0.5 * pow(a + alpha, -0.5f);
    return 0.5 * 1.0/sqrt(a + alpha);
}


__global__ void mat_sqrt_d_kernel (const float * __restrict__ src,
                                float * __restrict__ dst, int m, int n, float alpha){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if (row < m && col < n){
        dst[row * n + col] = mat_sqrt_d(src[row * n + col], alpha);
    }
}

void mat_sqrt_d_kernel_exec(const float *src, float *dst, int m, int n, float alpha){
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((n+block.x-1)/block.x, (m+block.y-1)/block.y);

    /* lunch kernel */
    mat_sqrt_d_kernel<<<grid, block>>>(src, dst, m, n, alpha);
    hipDeviceSynchronize();

}
