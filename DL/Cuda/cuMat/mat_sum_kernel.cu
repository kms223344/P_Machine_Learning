#include "hip/hip_runtime.h"

#include "mat_sum_kernel.h"

#define BLOCK_SIZE 32

__global__ void mat_sum_kernel (const float * __restrict__ src,
                                float * __restrict__ dst, int m, int n){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if (row < m && col < n){
        atomicAdd(dst, src[row * n + col]);
    }
}

void mat_sum_kernel_exec(const float *src, float *dst, int m, int n){
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((n+block.x-1)/block.x, (m+block.y-1)/block.y);

    /* lunch kernel */
    mat_sum_kernel<<<grid, block>>>(src, dst, m, n);
    hipDeviceSynchronize();
}
