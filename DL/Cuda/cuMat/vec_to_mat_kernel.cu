#include "hip/hip_runtime.h"

#include "vec_to_mat_kernel.h"

#define BLOCK_SIZE 32

__global__ void vec_to_mat_kernel (const float * __restrict__ src,
                                float * __restrict__ dst, int m, int n){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if (row < m && col < n){

        dst[row * n + col] = src[col];

    }
}

void vec_to_mat_kernel_exec(const float *src, float *dst, int m, int n){
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((n+block.x-1)/block.x, (m+block.y-1)/block.y);

    /* lunch kernel */
    vec_to_mat_kernel<<<grid, block>>>(src, dst, m, n);
    hipDeviceSynchronize();
}
