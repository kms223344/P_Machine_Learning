#include "hip/hip_runtime.h"
#include "dropout_kernel.h"
#include <hiprand/hiprand_kernel.h>
#include <windows.h>

#define BLOCK_SIZE 32

__device__ int WangHash(int a) {
    a = (a ^ 61) ^ (a >> 16);
    a = a + (a << 3);
    a = a ^ (a >> 4);
    a = a * 0x27d4eb2d;
    a = a ^ (a >> 15);
    return a;
}

__global__ void dropout_kernel (const float * __restrict__ src,
                                float * __restrict__ dst, float * __restrict__ dst_idx, int m, int n, float p, int seed){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if (row < m && col < n){

        // hiprand_init is very slow.
        // so we use the technique as bellow.
        // http://richiesams.blogspot.jp/2015/03/creating-randomness-and-acummulating.html
        // or https://devtalk.nvidia.com/default/topic/480586/hiprand-initialization-time/
        //generate random number
        int SEED = WangHash(seed);
        hiprandState_t state;
        int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
        //hiprand_init( (SEED << 20) + threadId, 0, 0, &state);
        hiprand_init( SEED + threadId, 0, 0, &state);
        float randNum = hiprand_uniform(&state);


        float scale = 1.0/(1.0-p);
        float flag = randNum >= p ? 1.0:0.0;
        /*
        if (randNum >= p){
            dst[row * n + col] = src[row * n + col] / (1.0-p);
            dst_idx[row * n + col] = 1.0f;
        }
        else{
            dst[row * n + col] = 0.0f;
            dst_idx[row * n + col] = 0.0f;
        }
        */
        float mask = scale * flag;
        dst_idx[row * n + col] = mask;
        dst[row * n + col] = src[row * n + col] * mask;

    }

}


//From->:https://stackoverflow.com/questions/5404277/porting-clock-gettime-to-windows
struct timespec2 { long tv_sec; long tv_nsec; };   //header part
#define exp7           10000000i64     //1E+7     //C-file part
#define exp9         1000000000i64     //1E+9
#define w2ux 116444736000000000i64     //1.jan1601 to 1.jan1970
void unix_time(struct timespec2* spec)
{
    __int64 wintime; GetSystemTimeAsFileTime((FILETIME*)&wintime);
    wintime -= w2ux;  spec->tv_sec = wintime / exp7;
    spec->tv_nsec = wintime % exp7 * 100;
}
int clock_gettime(int, timespec2* spec)
{
    static  struct timespec2 startspec; static double ticks2nano;
    static __int64 startticks, tps = 0;    __int64 tmp, curticks;
    QueryPerformanceFrequency((LARGE_INTEGER*)&tmp); //some strange system can
    if (tps != tmp) {
        tps = tmp; //init ~~ONCE         //possibly change freq ?
        QueryPerformanceCounter((LARGE_INTEGER*)&startticks);
        unix_time(&startspec); ticks2nano = (double)exp9 / tps;
    }
    QueryPerformanceCounter((LARGE_INTEGER*)&curticks); curticks -= startticks;
    spec->tv_sec = startspec.tv_sec + (curticks / tps);
    spec->tv_nsec = startspec.tv_nsec + (double)(curticks % tps) * ticks2nano;
    if (!(spec->tv_nsec < exp9)) { spec->tv_sec++; spec->tv_nsec -= exp9; }
    return 0;
}
//<-to:https://stackoverflow.com/questions/5404277/porting-clock-gettime-to-windows

void dropout_kernel_exec(const float *src, float *dst, float *dst_idx, int m, int n, float p){
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((n+block.x-1)/block.x, (m+block.y-1)/block.y);

    //int seed = time(0);
    struct timespec2 tm;
    clock_gettime(0, &tm);
    int seed = tm.tv_nsec;

    /* lunch kernel */
    dropout_kernel<<<grid, block>>>(src, dst, dst_idx, m, n, p, seed);
    hipDeviceSynchronize();
}
