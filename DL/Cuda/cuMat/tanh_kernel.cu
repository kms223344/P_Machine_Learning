#include "hip/hip_runtime.h"
#include "tanh_kernel.h"

#define BLOCK_SIZE 32

__device__ __forceinline__ float tanh_f(float a){
    return std::tanh(a);
}

__global__ void tanh_kernel (const float * __restrict__ src,
                                float * __restrict__ dst, int m, int n){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if (row < m && col < n){
        dst[row * n + col] = tanh_f(src[row * n + col]);
    }

}
void tanh_kernel_exec(const float *src, float *dst, int m, int n){
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((n+block.x-1)/block.x, (m+block.y-1)/block.y);

    /* lunch kernel */
    tanh_kernel<<<grid, block>>>(src, dst, m, n);
    hipDeviceSynchronize();
}
