#include "hip/hip_runtime.h"

#include "batch_sum_kernel.h"

#define BLOCK_SIZE 32

__global__ void batch_sum_kernel (const float * __restrict__ src,
                                float * __restrict__ dst, int m, int n){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if (row < m && col < n){
        atomicAdd(&dst[col], src[row * n + col]);
    }
}

void batch_sum_kernel_exec(const float *src, float *dst, int m, int n){
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((n+block.x-1)/block.x, (m+block.y-1)/block.y);

    /* lunch kernel */
    batch_sum_kernel<<<grid, block>>>(src, dst, m, n);
    hipDeviceSynchronize();
}
