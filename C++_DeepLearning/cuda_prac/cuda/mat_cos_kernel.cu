#include "hip/hip_runtime.h"
#include "mat_cos_kernel.h"

#define BLOCK_SIZE 32

__device__ __forceinline__ float mat_cos(float a, float alpha){
    return std::cos(a+alpha);
}

__global__ void mat_cos_kernel (const float * __restrict__ src,
                                float * __restrict__ dst, int m, int n, float alpha){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if (row < m && col < n){
        dst[row * n + col] = mat_cos(src[row * n + col], alpha);
    }

}

void mat_cos_kernel_exec(const float *src, float *dst, int m, int n, float alpha){
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((n+block.x-1)/block.x, (m+block.y-1)/block.y);

    /* lunch kernel */
    mat_cos_kernel<<<grid, block>>>(src, dst, m, n, alpha);
    hipDeviceSynchronize();

}
