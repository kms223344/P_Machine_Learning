#include "hip/hip_runtime.h"
#include "mat_inverse_d_kernel.h"

#define BLOCK_SIZE 32

__device__ __forceinline__ float mat_inverse_d (float a){
    return -1.0/((a+1e-8)*(a+1e-8));
}


__global__ void mat_inverse_d_kernel (const float * __restrict__ src,
                                float * __restrict__ dst, int m, int n){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if (row < m && col < n){
        dst[row * n + col] = mat_inverse_d(src[row * n + col]);
    }
}

void mat_inverse_d_kernel_exec(const float *src, float *dst, int m, int n){
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((n+block.x-1)/block.x, (m+block.y-1)/block.y);

    /* lunch kernel */
    mat_inverse_d_kernel<<<grid, block>>>(src, dst, m, n);
    hipDeviceSynchronize();

}
