#include "hip/hip_runtime.h"
#include "dropout_kernel.h"
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 32

__device__ int WangHash(int a) {
    a = (a ^ 61) ^ (a >> 16);
    a = a + (a << 3);
    a = a ^ (a >> 4);
    a = a * 0x27d4eb2d;
    a = a ^ (a >> 15);
    return a;
}

__global__ void dropout_kernel (const float * __restrict__ src,
                                float * __restrict__ dst, float * __restrict__ dst_idx, int m, int n, float p, int seed){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if (row < m && col < n){

        // hiprand_init is very slow.
        // so we use the technique as bellow.
        // http://richiesams.blogspot.jp/2015/03/creating-randomness-and-acummulating.html
        // or https://devtalk.nvidia.com/default/topic/480586/hiprand-initialization-time/
        //generate random number
        int SEED = WangHash(seed);
        hiprandState_t state;
        int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
        //hiprand_init( (SEED << 20) + threadId, 0, 0, &state);
        hiprand_init( SEED + threadId, 0, 0, &state);
        float randNum = hiprand_uniform(&state);


        float scale = 1.0/(1.0-p);
        float flag = randNum >= p ? 1.0:0.0;
        /*
        if (randNum >= p){
            dst[row * n + col] = src[row * n + col] / (1.0-p);
            dst_idx[row * n + col] = 1.0f;
        }
        else{
            dst[row * n + col] = 0.0f;
            dst_idx[row * n + col] = 0.0f;
        }
        */
        float mask = scale * flag;
        dst_idx[row * n + col] = mask;
        dst[row * n + col] = src[row * n + col] * mask;

    }

}

void dropout_kernel_exec(const float *src, float *dst, float *dst_idx, int m, int n, float p){
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((n+block.x-1)/block.x, (m+block.y-1)/block.y);

    //int seed = time(0);
    struct timespec tm;
    clock_gettime(CLOCK_REALTIME, &tm);
    int seed = tm.tv_nsec;

    /* lunch kernel */
    dropout_kernel<<<grid, block>>>(src, dst, dst_idx, m, n, p, seed);
    hipDeviceSynchronize();
}
